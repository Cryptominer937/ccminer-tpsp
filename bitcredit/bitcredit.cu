
extern "C"
{
#include "sph/sph_types.h"
#include "sph/sph_sha2.h"
#include "miner.h"
}

#include "cuda_helper.h"



static uint32_t *d_hash[MAX_GPUS] ;



extern void bitcredit_setBlockTarget(uint32_t * data,const uint32_t * midstate, const void *ptarget);
extern void bitcredit_cpu_init(uint32_t thr_id, int threads, uint32_t* hash);
extern uint32_t bitcredit_cpu_hash(uint32_t thr_id, int threads, uint32_t startNounce, int order);

 void credithash(void *state, const void *input)
{

	sph_sha256_context sha1,sha2;
	uint32_t hash[8],hash2[8];

	sph_sha256_init(&sha1);
	sph_sha256(&sha1, input, 168);
	sph_sha256_close(&sha1, hash);


	sph_sha256_init(&sha2);
	sph_sha256(&sha2, hash, 32);
	sph_sha256_close(&sha2, hash2);


	memcpy(state, hash2, 32);
}


     
extern "C" int scanhash_bitcredit(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, const uint32_t *midstate, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[35];
	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	const uint32_t Htarg = ptarget[7];
	int coef = 4;

	uint32_t throughput = 256*256*64*8;

	static bool init[MAX_GPUS] = { 0 };
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]); 
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		 
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id],  8 * sizeof(uint32_t) * throughput));
		bitcredit_cpu_init(thr_id, throughput, d_hash[thr_id]);
		init[thr_id] = true;
	}

	uint32_t endiandata[42],endianmid[8];
		for (int k = 0; k < 42; k++)
			be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	bitcredit_setBlockTarget(pdata,midstate,ptarget);
	uint64_t nloop = max_nonce/throughput + 1;
	do {
		int order = 0;
		uint32_t foundNonce = bitcredit_cpu_hash(thr_id, throughput, pdata[35], order++);
		if  (foundNonce != 0xffffffff)
		{

				pdata[35] = foundNonce;
				*hashes_done = foundNonce - first_nonce + 1;
				return 1;

		}
		if ((uint64_t)pdata[35] + throughput >(uint64_t)0xffffffff) {
                       pdata[35]=0xffffffff; 
                      *hashes_done = pdata[35] - first_nonce + 1; return 0;
        } else { 
    
		pdata[35] += throughput;}
	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);
	*hashes_done = pdata[35] - first_nonce + 1;
	return 0;
}
